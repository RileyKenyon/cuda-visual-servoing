#include "hip/hip_runtime.h"
///-----------------------------------------------------------------------------
/// @file imgProc.cu
///
/// @author Riley Kenyon (rike2277@colorado.edu)
/// @brief
///
/// @date 2024-01-27
///-----------------------------------------------------------------------------
#include "imgProc.hpp"
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

void convert_grayscale_cpu(cv::Mat *img, cv::Mat *grayscale) {
  for (int i = 0; i < img->rows * img->cols; i++) {
    grayscale[i] = img->data[3 * i] * 0.07 + img->data[3 * i + 1] * 0.72 + img->data[3 * i + 2] * 0.21;
  }
}

__global__ void gpu_grayscale(const unsigned char *mat, unsigned char *matG, int width, int height) {
  // Distance between array elements (i,j)[0] to (i,j)[1] is 1 not width*height
  int tid = blockIdx.x * blockDim.x + threadIdx.x; // thread ID
  int stride = blockDim.x * gridDim.x;             // stride lengths

  // grayscale calculation with strides
  while (tid < width * height) {
    matG[tid] = mat[3 * tid] * 0.07 + mat[3 * tid + 1] * 0.72 + mat[3 * tid + 2] * 0.21;
    tid = tid + stride;
  }
}

__global__ void screenAllocate(unsigned char *originalImage,
                               unsigned char *screenImage,
                               int *imageInfo,
                               int *screenInfo) {
  // Distance between array elements (i,j)[0] to (i,j)[1] is 1 not width*height
  // thread ID
  int imageWidth = imageInfo[0];
  int imageHeight = imageInfo[1];
  int screenX = screenInfo[0];
  int screenY = screenInfo[1];
  int screenWidth = screenInfo[2];
  int screenHeight = screenInfo[3];
  int index = 0;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  while (tid < screenWidth * screenHeight) {
    index = imageWidth * (screenY + tid / screenWidth) + screenX + (tid - screenWidth * (tid / screenWidth));
    screenImage[tid] = originalImage[index];
    tid = tid + stride;
  }
}
