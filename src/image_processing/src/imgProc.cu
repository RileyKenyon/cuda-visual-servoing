#include "hip/hip_runtime.h"
///-----------------------------------------------------------------------------
/// @file imgProc.cu
///
/// @author Riley Kenyon (rike2277@colorado.edu)
/// @brief
///
/// @date 2024-01-27
///-----------------------------------------------------------------------------
#include "imgProc.hpp"
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

void convert_grayscale_cpu(cv::Mat *img, cv::Mat *grayscale) {
  for (int i = 0; i < img->rows * img->cols; i++) {
    grayscale[i] = img->data[3 * i] * 0.07 + img->data[3 * i + 1] * 0.72 + img->data[3 * i + 2] * 0.21;
  }
}

__global__ void gpu_grayscale(const unsigned char *mat, unsigned char *matG, int width, int height) {
  // Distance between array elements (i,j)[0] to (i,j)[1] is 1 not width*height
  int tid = blockIdx.x * blockDim.x + threadIdx.x; // thread ID
  int stride = blockDim.x * gridDim.x;             // stride lengths

  // grayscale calculation with strides
  while (tid < width * height) {
    matG[tid] = mat[3 * tid] * 0.07 + mat[3 * tid + 1] * 0.72 + mat[3 * tid + 2] * 0.21;
    tid = tid + stride;
  }
}

__global__ void screenAllocate(const unsigned char *originalImage,
                               unsigned char *screenImage,
                               const int *imageInfo,
                               const int *screenInfo) {
  // Distance between array elements (i,j)[0] to (i,j)[1] is 1 not width*height
  // thread ID
  int imageWidth = imageInfo[0];
  int imageHeight = imageInfo[1];
  int screenX = screenInfo[0];
  int screenY = screenInfo[1];
  int screenWidth = screenInfo[2];
  int screenHeight = screenInfo[3];
  int index = 0;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  while (tid < screenWidth * screenHeight) {
    index = imageWidth * (screenY + tid / screenWidth) + screenX + (tid - screenWidth * (tid / screenWidth));
    screenImage[tid] = originalImage[index];
    tid = tid + stride;
  }
}

__global__ void edgeFind(const unsigned char *grayData,
                         unsigned char *edge,
                         int width,
                         int height,
                         int threshold = 140) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  while (tid < width * height) {
    if (tid > 3 * width) {
      if (grayData[tid] > threshold && grayData[tid - width] < threshold && grayData[tid - width - 1] < threshold &&
          grayData[tid - width + 1] < threshold && grayData[tid - 1] > threshold) { // probably easier way to do this
        edge[tid] = 255;                                                            // set to white
      } else {
        edge[tid] = 0;
      }
    }
    tid = tid + stride;
  }
}

__global__ void addArr(const unsigned char *arrA,
                       const unsigned char *arrB,
                       unsigned char *output,
                       int width,
                       int height) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  while (tid < width * height) {
    output[tid] = arrA[tid] + arrB[tid];
    tid = tid + stride;
  }
}