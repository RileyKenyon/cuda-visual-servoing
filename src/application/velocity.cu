#include "hip/hip_runtime.h"
#include "imgProc.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <string>

#define NUM 10000
#define Frames 120
// GPU KERNELS
//----------------------------------------------------------------
__global__ void addArr(unsigned char *arrA, unsigned char *arrB, unsigned char *output, int width, int height) {
  int tid, stride;
  tid = blockIdx.x * blockDim.x + threadIdx.x;
  stride = blockDim.x * gridDim.x;
  while (tid < width * height) {
    output[tid] = arrA[tid] + arrB[tid];
    tid = tid + stride;
  }
}

__global__ void spacing(unsigned char *pixelData, int width, int height) {
  int tid, stride, difference;
  tid = blockIdx.x * blockDim.x + threadIdx.x;
  stride = blockDim.x * gridDim.x;
  if (tid < width * height) {
    if (pixelData[tid] == 255) {
      int init = tid;
      tid = init + 10 * width;
      // looking below by up to 20 pixels
      while (tid < width * height && tid < init + width * 50) {
        if (255 - pixelData[tid] == 0) {
          difference = (tid - init) / width;
          printf("%d  ", difference);
          break;
        }
        tid = tid + width;
        difference = 0;
      }
    } else {
      difference = 0;
    }
  }
}
// MAIN FUNCTION
//-----------------------------------------------------------------
// using namespace cv;
int main() {
  // Initialize timer settings
  float calcTimer = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // float GPUtimer, CPUtimer;

  // ORIGINAL IMAGE
  //-------------------------------------------------------------------
  // Get initial image and print
  cv::Mat img;
  // img = cv::imread("pineapple.jpeg");
  cv::VideoCapture cap("color.avi");
  // cv::VideoCapture cap(1); // webcam
  if (!cap.isOpened()) {
    printf("Error getting Stream \n");
  }
  cap >> img;
  // cv::imshow("original",img);
  // cv::waitKey(0);
  int imageWidth = img.cols;
  int imageHeight = img.rows;
  printf("Resolution: %d x %d \n", imageWidth, imageHeight);
  // Do some ROI and calibration to select screen size
  cv::Rect2d r = cv::selectROI(img);
  int width = r.width;
  int height = r.height;
  // unsigned char *screenData = (unsigned char *) malloc(width*height*sizeof(unsigned char));

  unsigned int numThreads, numBlocksImage, numBlocksScreen;
  numThreads = 1024; // good number for multiple of 32
  numBlocksImage = (imageWidth * imageHeight + numThreads - 1) / numThreads;
  numBlocksScreen = (width * height + numThreads - 1) / numThreads;
  /** Write to file
    std::ofstream dataFile;
    dataFile.open ("output.txt");
    for (int k = 0; k<3; k++){
      for (int j = 100*(k+1)*width; j<((100*(k+1))+1)*width; j++){
         int output = lineData[j] - '0';
         dataFile << output;
         if (j != ((100*(k+1))+1)*width-1){ //last element in row
           dataFile << ",";
         }
         lineData[j] = 0;
      }
    dataFile << "\n";
    }
    dataFile.close();
    **/
  // SETUP SETTINGS
  //-------------------------------------------------------------------
  // Configure blocks and threads for GPU
  /**
  unsigned int numThreads, numBlocks;
  numThreads = 1024; // good number for multiple of 32
  numBlocks = (width*height + numThreads - 1)/numThreads;
  **/
  // Allocate device and host
  unsigned char *matA, *screenData, *grayData, *edge, *prevArr, *output;
  int *imageInfo, *screenInfo;
  hipMallocManaged(&matA, sizeof(unsigned char) * imageWidth * imageHeight * 3);
  hipMallocManaged(&grayData, sizeof(unsigned char) * imageWidth * imageHeight);
  hipMallocManaged(&screenData, sizeof(unsigned char) * width * height);
  hipMallocManaged(&edge, sizeof(unsigned char) * width * height);
  hipMallocManaged(&prevArr, sizeof(unsigned char) * width * height);
  hipMallocManaged(&output, sizeof(unsigned char) * width * height);
  hipMallocManaged(&imageInfo, sizeof(int) * 2);
  hipMallocManaged(&screenInfo, sizeof(int) * 4);

  // GPU CALCULATION
  //-----------------------------------------------------------------
  // Initial assignment to previous arr
  int imageInfoHost[2] = {imageWidth, imageHeight};
  int screenInfoHost[4] = {r.x, r.y, r.width, r.height};
  hipMemcpy(imageInfo, imageInfoHost, 2 * sizeof(int), hipMemcpyHostToDevice);   // FOR COPYING ARRAY
  hipMemcpy(screenInfo, screenInfoHost, 4 * sizeof(int), hipMemcpyHostToDevice); // FOR COPYING ARRAY
  printf("Size of image: %d, %d \n", imageInfo[0], imageInfo[1]);
  printf("Size of ROI: %d,%d,%d,%d \n", screenInfo[0], screenInfo[1], screenInfo[2], screenInfo[3]);
  hipMemcpy(matA,
             img.data,
             imageWidth * imageHeight * 3 * sizeof(unsigned char),
             hipMemcpyHostToDevice); // FOR COPYING ARRAY
  gpu_grayscale<<<numBlocksImage, numThreads>>>(matA, grayData, imageWidth, imageHeight);
  hipDeviceSynchronize();
  screenAllocate<<<numBlocksScreen, numThreads>>>(grayData, screenData, imageInfo, screenInfo);
  hipDeviceSynchronize();
  edgeFind<<<numBlocksScreen, numThreads>>>(screenData, prevArr, width, height);
  hipDeviceSynchronize();
  char c; // for waitkey
  cv::Mat test(cv::Size(width, height), CV_8UC1, screenData);
  cv::imshow("GPU", test);
  c = cv::waitKey(0);

  // Video Writer
  cv::VideoWriter writer;
  int codec = cv::VideoWriter::fourcc('M', 'J', 'P', 'G');
  double fps = 30;
  std::string filename = "./converted.avi";
  writer.open(filename, codec, fps, test.size(), 1); // boolean at end is color
  if (!writer.isOpened()) {
    printf("Unable to Open Video\n");
    return -1;
  }

  // Converter Loop
  for (;;) {
    hipEventRecord(start);
    for (int j = 0; j < Frames; j++) {
      // capture and calculations
      cap >> img;
      hipMemcpy(matA,
                 img.data,
                 imageWidth * imageHeight * 3 * sizeof(unsigned char),
                 hipMemcpyHostToDevice); // FOR COPYING ARRAY
      gpu_grayscale<<<numBlocksImage, numThreads>>>(matA, grayData, imageWidth, imageHeight);
      hipDeviceSynchronize(); // sync threads and cpy mem
      screenAllocate<<<numBlocksScreen, numThreads>>>(grayData, screenData, imageInfo, screenInfo);
      hipDeviceSynchronize();
      edgeFind<<<numBlocksScreen, numThreads>>>(screenData, edge, width, height);
      hipDeviceSynchronize();
      addArr<<<numBlocksScreen, numThreads>>>(edge, prevArr, output, width, height);
      hipDeviceSynchronize();
      spacing<<<numBlocksScreen, numThreads>>>(output, width, height);
      hipDeviceSynchronize();
      // std::cout << screenData;
      cv::Mat build(cv::Size(width, height), CV_8UC1, output);
      // cv::Mat build(cv::Size(imageWidth,imageHeight),CV_8UC1,grayData);
      memcpy(prevArr, edge, width * height * sizeof(unsigned char));

      // saving
      // cv::Mat videoFrameGray(cv::Size(width,height),CV_8UC3);
      cv::Mat videoFrameGray;
      cv::cvtColor(build, videoFrameGray, CV_GRAY2BGR); // 3 array of grayscale for saving to file
      // cv::imshow("savingExample",videoFrameGray);
      // c = cv::waitKey(0);
      writer.write(videoFrameGray); // write to video file

      cv::imshow("GPU", build);
      c = cv::waitKey(1);
      if (c == ' ')
        break;
    }
    if (c == ' ')
      break;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&calcTimer, start, stop);
    printf("FPS GPU: %f \n", Frames / calcTimer * 1000);
    calcTimer = 0;
  }
  // CLOSEOUT
  //--------------------------------------------------------------------------
  hipFree(matA);
  hipFree(grayData);
  hipFree(edge);
  hipFree(prevArr);
  hipFree(output);
}
