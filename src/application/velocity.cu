#include "imgProc.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <string>

#define NUM 10000
#define Frames 120
static constexpr unsigned int numThreads = 1024; // good number for multiple of 32

int main(int argc, char const *argv[]) {
  // Initialize timer settings
  float calcTimer = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // ORIGINAL IMAGE
  //-------------------------------------------------------------------
  // Get initial image and print
  cv::Mat img;
  cv::VideoCapture *cap = nullptr;
  if (argc == 2) {
    std::string fname = argv[1];
    if (fname.substr(fname.size() - 4, fname.size()) == ".avi") {
      cap = new cv::VideoCapture(fname);
      if (!cap->isOpened()) {
        std::runtime_error("Error getting Stream");
      }
      *cap >> img;
    } else if (fname.substr(fname.size() - 5, fname.size()) == ".jpeg") {
      img = cv::imread(fname);
    }
  }

  if (img.empty()) {
    cap = new cv::VideoCapture(1); // webcam
    if (!cap->isOpened()) {
      std::runtime_error("Error getting Stream");
    }
    *cap >> img;
  }
  cv::imshow("original", img);
  cv::waitKey(0);

  const int imageWidth = img.cols;
  const int imageHeight = img.rows;

  // Do some ROI and calibration to select screen size
  cv::Rect2d r = cv::selectROI(img);
  const int width = r.width;
  const int height = r.height;

  unsigned int numBlocksImage = (imageWidth * imageHeight + numThreads - 1) / numThreads;
  unsigned int numBlocksScreen = (width * height + numThreads - 1) / numThreads;

  // SETUP SETTINGS
  //-------------------------------------------------------------------
  // Allocate device and host
  unsigned char *matA, *screenData, *grayData, *edge, *prevArr, *output;
  int *imageInfo, *screenInfo;
  hipMallocManaged(&matA, sizeof(unsigned char) * imageWidth * imageHeight * 3);
  hipMallocManaged(&grayData, sizeof(unsigned char) * imageWidth * imageHeight);
  hipMallocManaged(&screenData, sizeof(unsigned char) * width * height);
  hipMallocManaged(&edge, sizeof(unsigned char) * width * height);
  hipMallocManaged(&prevArr, sizeof(unsigned char) * width * height);
  hipMallocManaged(&output, sizeof(unsigned char) * width * height);
  hipMallocManaged(&imageInfo, sizeof(int) * 2);
  hipMallocManaged(&screenInfo, sizeof(int) * 4);

  // GPU CALCULATION
  //-----------------------------------------------------------------
  // Initial assignment to previous arr
  int imageInfoHost[2] = {imageWidth, imageHeight};
  int screenInfoHost[4] = {r.x, r.y, r.width, r.height};
  hipMemcpy(imageInfo, imageInfoHost, 2 * sizeof(int), hipMemcpyHostToDevice);   // FOR COPYING ARRAY
  hipMemcpy(screenInfo, screenInfoHost, 4 * sizeof(int), hipMemcpyHostToDevice); // FOR COPYING ARRAY
  std::cout << "Size of image: " << imageInfo[0] << ", " << imageInfo[1] << std::endl;
  std::cout << "Size of ROI: " << screenInfo[0] << ", " << screenInfo[1] << ", " << screenInfo[2] << ", "
            << screenInfo[3] << std::endl;
  hipMemcpy(matA,
             img.data,
             imageWidth * imageHeight * 3 * sizeof(unsigned char),
             hipMemcpyHostToDevice); // FOR COPYING ARRAY
  gpu_grayscale<<<numBlocksImage, numThreads>>>(matA, grayData, imageWidth, imageHeight);
  hipDeviceSynchronize();
  screenAllocate<<<numBlocksScreen, numThreads>>>(grayData, screenData, imageInfo, screenInfo);
  hipDeviceSynchronize();
  edgeFind<<<numBlocksScreen, numThreads>>>(screenData, prevArr, width, height);
  hipDeviceSynchronize();
  char c; // for waitkey
  cv::Mat test(cv::Size(width, height), CV_8UC1, screenData);
  cv::imshow("GPU", test);
  c = cv::waitKey(0);

  // Video Writer
  cv::VideoWriter writer;
  int codec = cv::VideoWriter::fourcc('M', 'J', 'P', 'G');
  double fps = 30;
  std::string filename = "./converted.avi";
  writer.open(filename, codec, fps, test.size(), 1); // boolean at end is color
  if (!writer.isOpened()) {
    printf("Unable to Open Video\n");
    return -1;
  }

  // Converter Loop
  for (;;) {
    hipEventRecord(start);
    for (int j = 0; j < Frames; j++) {
      // capture and calculations
      if (cap != nullptr) {
        *cap >> img;
      }
      hipMemcpy(matA,
                 img.data,
                 imageWidth * imageHeight * 3 * sizeof(unsigned char),
                 hipMemcpyHostToDevice); // FOR COPYING ARRAY
      gpu_grayscale<<<numBlocksImage, numThreads>>>(matA, grayData, imageWidth, imageHeight);
      hipDeviceSynchronize(); // sync threads and cpy mem
      screenAllocate<<<numBlocksScreen, numThreads>>>(grayData, screenData, imageInfo, screenInfo);
      hipDeviceSynchronize();
      edgeFind<<<numBlocksScreen, numThreads>>>(screenData, edge, width, height);
      hipDeviceSynchronize();
      addArr<<<numBlocksScreen, numThreads>>>(edge, prevArr, output, width, height);
      hipDeviceSynchronize();
      spacing<<<numBlocksScreen, numThreads>>>(output, width, height);
      hipDeviceSynchronize();
      // std::cout << screenData;
      cv::Mat build(cv::Size(width, height), CV_8UC1, output);
      // cv::Mat build(cv::Size(imageWidth,imageHeight),CV_8UC1,grayData);
      memcpy(prevArr, edge, width * height * sizeof(unsigned char));

      // saving
      // cv::Mat videoFrameGray(cv::Size(width,height),CV_8UC3);
      cv::Mat videoFrameGray;
      cv::cvtColor(build, videoFrameGray, cv::COLOR_GRAY2BGR); // 3 array of grayscale for saving to file
      // cv::imshow("savingExample",videoFrameGray);
      // c = cv::waitKey(0);
      writer.write(videoFrameGray); // write to video file

      cv::imshow("GPU", build);
      c = cv::waitKey(1);
      if (c == ' ')
        break;
    }
    if (c == ' ')
      break;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&calcTimer, start, stop);
    printf("FPS GPU: %f \n", Frames / calcTimer * 1000);
    calcTimer = 0;
  }
  // CLOSEOUT
  //--------------------------------------------------------------------------
  hipFree(matA);
  hipFree(grayData);
  hipFree(edge);
  hipFree(prevArr);
  hipFree(output);
  if (cap != nullptr) {
    delete cap;
    cap = nullptr;
  }
}
