#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <string>

#define NUM 10000
#define Frames 120
// Global variables
/**
const int row_slider_max = 100;
const int col_slider_max = 100;
cv::Mat grab;
// Trackbar function

void trackbar1 ( int pos, void*){
  imshow("cpu_gray",grab)
}
**/
__global__ void gpu_grayscale(unsigned char *matA,unsigned char *grayData, int width, int height){
  //Distance between array elements (i,j)[0] to (i,j)[1] is 1 not width*height
  //thread ID
  int tid;
  tid = blockIdx.x*blockDim.x + threadIdx.x;
  
  //stride lengths
  int stride;
  stride = blockDim.x*gridDim.x;

  // grayscale calculation with strides
  while (tid < width*height){
    grayData[tid] = matA[3*tid]*0.07 + matA[3*tid+1]*0.72 + matA[3*tid+2]*0.21;
    if (grayData[tid] > 170 ){ //saturate to either 255 or 0 - for pixel testing
      grayData[tid] = 255;
    } else {
      grayData[tid] = 0;
    }
    tid = tid + stride;
  }
  
}
// MAIN FUNCTION
//-----------------------------------------------------------------
int main()
{
  // Initialize timer settings
  float calcTimer = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //float GPUtimer, CPUtimer;

// ORIGINAL IMAGE
//-------------------------------------------------------------------
  // Get initial image and print
  cv::Mat img;
  img = cv::imread("Screen.png");
  //cv::imshow("original",img);
  unsigned char *cpuData = (unsigned char *) malloc(img.cols*img.rows*sizeof(unsigned char));
  for (int i = 0; i < img.rows*img.cols; i++){
    cpuData[i] = img.data[3*i]*0.07 + img.data[3*i+1]*0.72 + img.data[3*i+2]*0.21;
  }
  cv::Mat reconstruction(cv::Size(img.cols,img.rows),CV_8UC1,cpuData);
  cv::imshow("cpu_gray",reconstruction);
  cv::waitKey(0);
  // select ROI
  cv::Rect2d r = cv::selectROI(reconstruction);
  //cv::Mat cropped = reconstruction(r);
  //cv::imshow("cropped",cropped);
  //cv::waitKey();
// CALCULATION SETTINGS
//-------------------------------------------------------------------  
  // Size of windows  
  int width = r.width;
  int height = r.height;
  // Configure blocks and threads for GPU
  unsigned int numThreads, numBlocks;
  numThreads = 1024; // good number for multiple of 32
  numBlocks = (width*height + numThreads - 1)/numThreads; // make so at maximum only one additional block

  // Allocate device and host
  unsigned char *matA, *grayData;
  hipMallocManaged(&matA,sizeof(unsigned char)*width*height*3);
  hipMallocManaged(&grayData,sizeof(unsigned char)*width*height);
  unsigned char *lineData = (unsigned char *) malloc(width*height*sizeof(unsigned char));

  //Edge detection
  cv::Mat kernel = (cv::Mat_<int>(3,2) <<
      1,1,
      0,0,
      0,0); 
  cv::Mat outputGPU;

/**
// GPU CALCULATION
//-----------------------------------------------------------------
  //hipEventRecord(start);
  for (;;){
    hipEventRecord(start);
    char c;
    for (int j = 0; j< Frames; j++){
      cap >> img;
      hipMemcpy(matA, img.data, width*height*3* sizeof(unsigned char), hipMemcpyHostToDevice); // NEED THIS LINE FOR COPYING ARRAY
      gpu_grayscale<<<numBlocks,numThreads>>>(matA,grayData,width,height);
      hipDeviceSynchronize(); // sync threads and transfer memory
    // Show grayscale image
      //cv::Mat outputGPU(cv::Size(width,height),CV_8UC1,grayData);
      cv::Mat build(cv::Size(width,height),CV_8UC1,grayData); // NEED TO CHANGE TO GPU DATA IF USING CUDAMALLOC AND NOT CUDAMALLOCMANAGED
      // implement edge detect
      cv::morphologyEx(build,outputGPU,cv::MORPH_HITMISS,kernel);  
      //cv::rotate(outputGPU,outputGPU,cv::ROTATE_90_COUNTERCLOCKWISE); // for rotating recorded video    
      cv::imshow("GPU",outputGPU);
      c = cv::waitKey(1);
      if (c ==' ')
        break;
    }
    if (c ==' ')
      break;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&calcTimer,start, stop);
    printf("FPS GPU: %f \n",Frames/calcTimer*1000);
    calcTimer = 0;
  }
**/
// CPU CALCULATION
//--------------------------------------------------------------------
  //for (;;){
    hipEventRecord(start);
    char c;
    //for (int j = 0; j<Frames; j++){
    for (int y =r.y; y < (r.y + r.height); y++){
      //printf("-----------------------------------------------------\n");
      for (int x = r.x; x< (r.x + r.width); x++){
      int index = (y*width+x) - (r.y*width + r.x);
      int roi = y*img.cols + x;
      //printf("new Mat: %d Old Mat: %d\n",index,roi);
      printf("width: %g height: %g\n",r.width,r.height);
      printf("y_initial: %g x_initial: %g\n",r.y,r.x);
      lineData[index] = reconstruction.data[roi]; // cropped version of the original image with roi
      }
    }
    // Write to file
    std::ofstream dataFile;
    dataFile.open ("output.txt");
    for (int k = 0; k<3; k++){
      for (int j = 100*(k+1)*width; j<((100*(k+1))+1)*width; j++){
         int output = lineData[j] - '0';
         dataFile << output;
         if (j != ((100*(k+1))+1)*width-1){ //last element in row
           dataFile << ",";
         }
         lineData[j] = 0;
      }
    dataFile << "\n";
    }
    dataFile.close();
    
    // Create calibration lines
    for( int k = 0; k<3; k++){
      for(int j = 0; j < (r.height); j++){
        int number = r.width*j + 100*(k+1); // offset by 100  
	lineData[number] = 0;        
      }    
    }

    // Display Image
    cv::Mat grab(cv::Size(width,height),CV_8UC1,lineData);
    cv::imshow("cpu_gray",grab);
    cv::waitKey(0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&calcTimer,start, stop);
    printf("FPS CPU: %f\n",Frames/calcTimer*1000);
    calcTimer = 0;
  //}
// CLOSEOUT
//--------------------------------------------------------------------------
  //hipFree(matA);
  //hipFree(grayData);
  free(lineData);
  free(cpuData);
  return 0;
}
