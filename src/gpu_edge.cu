#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <string>

#define NUM 10000
#define Frames 120
//GPU KERNELS
//----------------------------------------------------------------
__global__ void gpu_grayscale(unsigned char *matA,unsigned char *grayData, int width, int height){
  //Distance between array elements (i,j)[0] to (i,j)[1] is 1 not width*height
  //thread ID
  int tid,stride;
  tid = blockIdx.x*blockDim.x + threadIdx.x;
  stride = blockDim.x*gridDim.x;

  while (tid < width*height){
    grayData[tid] = matA[3*tid]*0.07 + matA[3*tid+1]*0.72 + matA[3*tid+2]*0.21;
  tid = tid + stride;
  }
}
__global__ void edgeFind(unsigned char *grayData, unsigned char *edge, int width, int height){
  int tid, stride;
  tid = blockIdx.x*blockDim.x + threadIdx.x;
  stride = blockDim.x*gridDim.x;

  while (tid < width*height){
    if (tid > 2*width){
      if (grayData[tid] > 150 && grayData[tid-width] < 150 && grayData[tid-2*width] < 150 && grayData[tid-2] > 150 && grayData[tid-1] > 150){
        edge[tid] = 255; // set to white
      } else {
        edge[tid] = 0;
      }
    }
  tid = tid + stride;
  }
}
// MAIN FUNCTION
//-----------------------------------------------------------------
//using namespace cv;
int main()
{
  // Initialize timer settings
  float calcTimer = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //float GPUtimer, CPUtimer;

// ORIGINAL IMAGE
//-------------------------------------------------------------------
  // Get initial image and print
  cv::Mat img;
  //img = cv::imread("pineapple.jpeg");
  //cv::VideoCapture cap("example.mp4"); // replace with 1 if using webcam
  cv::VideoCapture cap(1);
  if (!cap.isOpened()){
    printf("Error getting Stream \n");
  }
  cap >> img;
  cv::imshow("original",img);
  cv::waitKey();  
  int width = img.cols;
  int height = img.rows;
  printf("Resolution: %d x %d \n",width, height);

// SETUP SETTINGS
//-------------------------------------------------------------------  
  // Configure blocks and threads for GPU
  unsigned int numThreads, numBlocks;
  numThreads = 1024; // good number for multiple of 32
  numBlocks = (width*height + numThreads - 1)/numThreads;

  // Allocate device and host
  unsigned char *matA, *grayData, *edge;
  hipMallocManaged(&matA,sizeof(unsigned char)*width*height*3);
  hipMallocManaged(&grayData,sizeof(unsigned char)*width*height);
  hipMallocManaged(&edge,sizeof(unsigned char)*width*height);

  // Video Writer
  cv::VideoWriter writer;
  int codec = cv::VideoWriter::fourcc('M','J','P','G');
  double fps = 30;
  std::string filename = "./example.avi";
  writer.open(filename,codec,fps,img.size(),1);
  if (!writer.isOpened()){
    printf("Unable to Open Video\n");
    return -1;
  }
// GPU CALCULATION
//-----------------------------------------------------------------
  //hipEventRecord(start);
  for (;;){
    hipEventRecord(start);
    char c;
    for (int j = 0; j< Frames; j++){
      cap >> img;
      hipMemcpy(matA, img.data, width*height*3* sizeof(unsigned char), hipMemcpyHostToDevice); // NEED THIS LINE FOR COPYING ARRAY
      gpu_grayscale<<<numBlocks,numThreads>>>(matA,grayData,width,height);
      hipDeviceSynchronize(); // sync threads and transfer memory
      // Edge Find
      edgeFind<<<numBlocks,numThreads>>>(grayData,edge,width,height);
      hipDeviceSynchronize();
      cv::Mat build(cv::Size(width,height),CV_8UC1,edge);    
      cv::Mat videoFrameGray; // 3 array of grayscale for saving to file
      cv::cvtColor(build,videoFrameGray,CV_GRAY2BGR);
      writer.write(videoFrameGray);  // write to video file
      cv::imshow("GPU",build);
      c = cv::waitKey(1);
      if (c ==' ')
        break;
    }
    if (c ==' ')
      break;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&calcTimer,start, stop);
    printf("FPS GPU: %f \n",Frames/calcTimer*1000);
    calcTimer = 0;

  }

// CLOSEOUT
//--------------------------------------------------------------------------
  hipFree(matA);
  hipFree(grayData);
}
